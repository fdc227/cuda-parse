#include "hip/hip_runtime.h"
#include "utilities.cuh"


#define num 31
#define totalwidth 500
#define xwidth 50
#define elenum 11
#define timesteps 5001

using namespace std;

void vectorcpy(double* a, std::vector<double>& vec, size_t size)
{
    for(size_t i = 0; i < size; i ++)
    {
        a[i] = vec[i];
    }
}

__global__ void gpu_compute(double* y_d, double* y_dot_d, double* y_torsion_d, double* X, double* y_gen, double* torsion_gen) // 500 threads per block, per thread calculates one value for y and torsion
{
    int M = threadIdx.x / xwidth;
    int N = threadIdx.x % xwidth;
    int blockidx = blockDim.x * blockIdx.x;
    int blockidy = blockDim.y * blockIdx.y; // representing time
    int index_x = blockidx + threadIdx.x;
    int index_y = blockidy + threadIdx.y;

    int total_index = index_x + totalwidth * index_y;

    __shared__ double y_local[elenum];
    __shared__ double y_dot_local[elenum];
    __shared__ double y_torsion_local[elenum];
    __shared__ double X_local[xwidth];

    if (threadIdx.x < elenum * 3 + xwidth)
    {
        if (threadIdx.x < elenum)
        {
            y_local[threadIdx.x] = y_d[elenum * index_y + threadIdx.x];
            if (blockIdx.x == 0)
            {
                printf("ylocal[%d]=%f", threadIdx.x, y_d[elenum * index_y + threadIdx.x]);
                /*cout << "y_local[" << threadIdx.x << "] = " << y_d[elenum * index_y + threadIdx.x] << "copied into cache" << endl;;*/
            }
        }
        else if (threadIdx.x < elenum * 2)
        {
            y_dot_local[threadIdx.x - elenum] = y_dot_d[elenum * index_y + threadIdx.x - elenum];
 /*           if (blockIdx.x == 0)
            {
                cout << "y_dot_local[" << threadIdx.x - elenum << "] = " << y_dot_d[elenum * index_y + threadIdx.x - elenum] << "copied into cache" << endl;;
            }*/
        }
        else if (threadIdx.x < elenum * 3)
        {
            y_torsion_local[threadIdx.x - elenum * 2] = y_torsion_d[elenum * index_y + threadIdx.x - 2 * elenum];
            //if (blockIdx.x == 0)
            //{
            //    cout << "y_torsion_local[" << threadIdx.x - elenum*2 << "] = " << y_torsion_d[elenum * index_y + threadIdx.x - 2 * elenum] << "copied into cache" << endl;;
            //}
        }
        else
        {
            X_local[threadIdx.x - elenum * 3] = X[threadIdx.x - elenum * 3];
      /*      if (blockIdx.x == 0)
            {
                cout << "X_local[" << threadIdx.x - elenum * 3 << "] = " << X[threadIdx.x - elenum * 3] << "copied into cache" << endl;;
            }*/
        }
    }
    __syncthreads();

    double q0 = y_local[M];
    double q0_dot = y_dot_local[M];
    double q0t = y_torsion_local[M];
    double q1 = y_local[M + 1];
    double q1_dot = y_dot_local[M + 1];
    double q1t = y_torsion_local[M + 1];
    double x = X_local[N];

    double y, torsion;

    y = q0 * (1 - 3 * pow(x, 2) + 2 * pow(x, 3)) + q0_dot * (x - 2 * pow(x, 2) + pow(x, 3)) + q1 * (3 * pow(x, 2) - 2 * pow(x, 3)) + q1_dot * (-pow(x, 2) + pow(x, 3));
    torsion = q0t * (1 - x) + q1t * x;

    y_gen[total_index] = y;
    torsion_gen[total_index] = torsion;
}

__global__ void gpu_compute_simple(double* y_d, double* y_dot_d, double* y_torsion_d, double* X, double* y_gen, double* torsion_gen)
{
    int array_idx = elenum * blockIdx.y + threadIdx.x;

    double q0 = y_d[array_idx];
    double q1 = y_d[array_idx + 1];
    double q0_dot = y_dot_d[array_idx];
    double q1_dot = y_dot_d[array_idx + 1];
    double qt0 = y_torsion_d[array_idx];
    double qt1 = y_torsion_d[array_idx + 1];

    int out_idx = xwidth *(elenum - 1) * blockIdx.y + xwidth * threadIdx.x;
    for (int i = 0; i < xwidth; i++)
    {
        double x = X[i];
        double y, torsion;
        y = q0 * (1 - 3 * pow(x, 2) + 2 * pow(x, 3)) + q0_dot * (x - 2 * pow(x, 2) + pow(x, 3)) + q1 * (3 * pow(x, 2) - 2 * pow(x, 3)) + q1_dot * (-pow(x, 2) + pow(x, 3));
        torsion = qt0 * (1 - x) + qt1 * x;
        y_gen[out_idx + i] = y;
        torsion_gen[out_idx + i] = torsion;
    }
}

int main(void)
{
    string filename = "sol.dat";
    ifstream file;
    file.open(filename, ios::in);
    vector<double> t;
    vector<double> y;
    vector<double> y_dot;
    vector<double> y_torsion;
    double local;

DATA:
    {
        for (int i = 0; i < num; i++)
        {
            if (!file.eof())
            {
                file >> local;
                if (i == 0)
                {
                    t.push_back(local);
                    y.push_back(0.0);
                    y_dot.push_back(0.0);
                    y_torsion.push_back(0.0);
                }
                else if ((i - 1) % 3 == 0)
                {
                    y.push_back(local);
                }
                else if ((i - 1) % 3 == 1)
                {
                    y_dot.push_back(local);
                }
                else
                {
                    y_torsion.push_back(local);
                }

                if (i == num - 1)
                    goto DATA;
            }
            else
                goto EXIT;
        }
    }
EXIT:

 /*   cout << t.size() << endl;
    cout << y.size() << endl;
    cout << y_dot.size() << endl;
    cout << y_torsion.size() << endl;

    cout << t.back() << endl;
    cout << y.back() << endl;
    cout << y_dot.back() << endl;
    cout << y_torsion.back() << endl;*/

    size_t yn = y.size();
    size_t tn = y_torsion.size();

    // Host vectors
    double* t_h, * y_h, * y_dot_h, * y_torsion_h, * X_h, * y_gen_h, * y_torsion_gen_h;
    t_h = new double[t.size()];
    y_h = new double[y.size()];
    y_dot_h = new double[y_dot.size()];
    y_torsion_h = new double[y_torsion.size()];
    X_h = new double[xwidth];
    y_gen_h = new double[(elenum - 1) * xwidth * timesteps];
    y_torsion_gen_h = new double[(elenum - 1) * xwidth * timesteps];


    vectorcpy(t_h, t, t.size());
    vectorcpy(y_h, y, y.size());
    vectorcpy(y_dot_h, y_dot, y_dot.size());
    vectorcpy(y_torsion_h, y_torsion, y_torsion.size());

    write_array_to_file(y_h, "y_h.dat", 5001, 11);
    write_array_to_file(y_dot_h, "y_dot_h.dat", 5001, 11);
    write_array_to_file(y_torsion_h, "y_torsion_h.dat", 5001, 11);

    cout << y_h[0] << endl;
    cout << y_dot_h[0] << endl;
    cout << y_torsion_h[0] << endl;

    cout << y_h[yn-1] << endl;
    cout << y_dot_h[yn-1] << endl;
    cout << y_torsion_h[yn-1] << endl;



    for (int i = 0; i < xwidth; i++)
    {
        X_h[i] = 1.0 / double(xwidth) * double(i);
    }

    void* host_array[6] = { (void*)y_h, (void*)y_dot_h, (void*)y_torsion_h, (void*)X_h, (void*)y_gen_h, (void*)y_torsion_gen_h };
    void* host_array2[4] = {(void*)y_h, (void*)y_dot_h, (void*)y_torsion_h, (void*)X_h };


    double *y_d, *y_dot_d, *y_torsion_d, *X_d, *y_gen_d, *y_torsion_gen_d;
    void** device_array[6] = {(void**)&y_d, (void**)&y_dot_d, (void**)&y_torsion_d, (void**)&X_d, (void**)&y_gen_d, (void**)&y_torsion_gen_d};
    void** device_array2[4] = { (void**)&y_d, (void**)&y_dot_d, (void**)&y_torsion_d, (void**)&X_d };
    size_t size[6] = { yn * sizeof(double), yn * sizeof(double), yn * sizeof(double), xwidth * sizeof(double), (elenum - 1) * xwidth * timesteps * sizeof(double), (elenum - 1) * xwidth * timesteps * sizeof(double) };
    size_t size2[4] = { yn * sizeof(double), yn * sizeof(double), yn * sizeof(double), xwidth * sizeof(double) };

    hipError_t cudastatus;
    cudastatus = arrayMalloc(device_array, 6, size);
    cudastatus = arraycpyHtoD_v2(device_array2, host_array2, 4, size2);

    dim3 DimGrid(1, timesteps, 1);
    dim3 DimBlock((elenum-1), 1, 1);
    gpu_compute_simple <<<DimGrid, DimBlock >>> (y_d, y_dot_d, y_torsion_d, X_d, y_gen_d, y_torsion_gen_d) ;

    cudastatus = onecpyDtoH(y_gen_h, y_gen_d, (elenum - 1) * xwidth * timesteps * sizeof(double));
    cudastatus = onecpyDtoH(y_torsion_gen_h, y_torsion_gen_d, (elenum - 1) * xwidth * timesteps * sizeof(double));

    if (cudastatus != hipSuccess)
    {
        fstream errfile;
        errfile.open("gpu_log.txt", ios::out);
        errfile << hipGetErrorString(cudastatus) << endl;
    }

    /*while (cin.get() != 'q')
    {
        int n = cin.get();
        cout << "y[" << n << "] is " << y_gen_h[n] << endl;
        cout << "torsion[" << n << "] is " << y_torsion_gen_h[n] << endl;
    }*/

    //vector<double> Y(y_gen_h, y_gen_h + (elenum - 1) * xwidth * timesteps);
    //vector<double> T(y_torsion_gen_h, y_torsion_gen_h + (elenum - 1) * xwidth * timesteps);

    write_array_to_file(y_gen_h, "y.dat", timesteps, (elenum - 1)* xwidth);
    write_array_to_file(y_torsion_gen_h, "torsion.dat", timesteps, (elenum - 1)* xwidth);

    return 0;
}
